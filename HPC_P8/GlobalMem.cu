#include "hip/hip_runtime.h"
%%writefile FunctionGlobal.cu
#include<stdio.h>
#include<hip/hip_runtime.h>

__managed__ int a[5]; 

__global__ void funGlobal()    //kernel definition
{
    for(int i=0;i<5;i++)
    {
        printf("%d\t", a[i]);
        a[i]=a[i]*5;
    }
}

int main()
{
    for(int i=0;i<5;i++)
        a[i]=0+i;

    funGlobal<<<1,1>>>( ); 

    hipDeviceSynchronize();

    printf("\n");
    for(int i=0;i<5;i++)
    {
        printf("%d\t",a[i]);
    }

    return 0;
}
