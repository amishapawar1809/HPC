#include "hip/hip_runtime.h"
%%writefile SumOfVectorsPrivate.cu
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorsadd(int *x,int *y, int *z)    //kernel definition
{
    int index=threadIdx.x; 
    int p=x[index]+y[index];
    z[index]=p;
}

int main()
{
    int n=10;
    int a[n], b[n],c[n];
    int *d,*e,*f;
    //printf("\n Enter elements of first array\n");
    for(int i=0;i<n;i++)
    {
        //scanf("%d",&a[i]);
        a[i]=rand()%10;
    }
    //printf("\n Enter elements of second array\n");
    for(int i=0;i<n;i++)
    {
        //scanf("%d",&b[i]);
        b[i]=rand()%10;
    }

    hipMalloc((void **)&d,n*sizeof(int));
    hipMalloc((void **)&e,n*sizeof(int));
    hipMalloc((void **)&f,n*sizeof(int));

    hipMemcpy(d,a,n*sizeof(int),hipMemcpyHostToDevice);   
    hipMemcpy(e,b,n*sizeof(int),hipMemcpyHostToDevice);
 
    vectorsadd<<<n,10>>>(d,e,f); 

    hipMemcpy(c,f,n*sizeof(int),hipMemcpyDeviceToHost);
    
    printf("\nElements of 1st array:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t",a[i]);
    }

    printf("\nElements of 2nd array:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t",b[i]);
    }

    printf("\nSum of two arrays:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t",c[i]);
    }

    /* Free the memory allocated to pointers d,e,f */
    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
