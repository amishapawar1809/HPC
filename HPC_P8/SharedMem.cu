#include "hip/hip_runtime.h"
%%writefile MatrixMultiplicationShared.cu
#include<stdio.h>
#include<hip/hip_runtime.h>
const int row1=3, col1=3, row2=3, col2=3;

__global__ void matrixMulShared(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int k=threadIdx.x;
 
    __shared__ int p[col1];

    n[col2*y+x]=0;

    p[k]=l[col1*y+k]*m[col2*k+x];

    __syncthreads();

    for(int i=0;i<col1;i++)
        n[col2*y+x]=n[col2*y+x]+p[i];
}

int main()
{
    int a[row1][col1];
    int b[row2][col2];
    int c[row1][col2];
    int *dev_a,*dev_b,*dev_c;

    //printf("\n Enter elements of first matrix of size "+ %d + "*" + %d + "\n", row1, col1);
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col1;j++)
        {
            //scanf("%d",&a[i][j]);
            a[i][j]=rand()%10;
        }
    }
    //printf("\n Enter elements of second matrix of size "+ %d + "*" + %d + "\n", row2, col2);
    for(int i=0;i<row2;i++)
    {
        for(int j=0;j<col2;j++)
        {
            //scanf("%d",&b[i][j]);
            b[i][j]=rand()%10;
        }
    }

    hipMalloc((void **)&dev_a,row1*col1*sizeof(int));
    hipMalloc((void **)&dev_b,row2*col2*sizeof(int));
    hipMalloc((void **)&dev_c,row1*col2*sizeof(int));

    hipMemcpy(dev_a,a,row1*col1*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,row2*col2*sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(col2,row1);

    matrixMulShared<<<grid,col1>>>(dev_a,dev_b,dev_c);

    hipMemcpy(c,dev_c,row1*col2*sizeof(int),hipMemcpyDeviceToHost);

    printf("\nElements of first matrix of size %d * %d \n", row1, col1);
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col1;j++)
        {
            printf("%d\t",a[i][j]);
        }
        printf("\n");
    }
 
    printf("\nElements of second matrix of size  %d * %d \n", row2, col2);
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col1;j++)
        {
            printf("%d\t",b[i][j]);
        }
        printf("\n");  
    }

    printf("\nProduct of two matrices:\n");
    for(int i=0;i<row1;i++)
    {
        for(int j=0;j<col2;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
