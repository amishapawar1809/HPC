#include "hip/hip_runtime.h"
%%writefile fibonacci.cu
#include<stdio.h>
#include<hip/hip_runtime.h>


// Fibonacci numbers upto n = 5
 //int f[6] = { 0, 1, 1, 2, 3, 5 };

__global__ void fiboSeries(int *a, int n)
{
    double PHI = 1.6180339;
    int f[6] = { 0, 1, 1, 2, 3, 5 };
    int index=blockIdx.x;
    if (index < 6){
        a[index]=f[index];
   }
    else{
     int t = 5, fn = 5;

     while (t < index) {
         fn = round(fn * PHI);
         t++;
     }
     a[index]=fn;
  }


}

int main()
{
    int n=10;
    int *a;
    int *dev_a;

    a = (int *)malloc(n * sizeof(int));

    hipMalloc((void **)&dev_a, n*sizeof(int));

    hipMemcpy(dev_a, a, n*sizeof(int), hipMemcpyHostToDevice);

    fiboSeries<<<n,1>>>(dev_a, n);

    hipMemcpy(a, dev_a, n*sizeof(int), hipMemcpyDeviceToHost);

    printf("\nFibonnaci Series: ");
    for(int i=0;i<n;i++)
       printf("%d ",a[i]);

    hipFree(dev_a);

}
