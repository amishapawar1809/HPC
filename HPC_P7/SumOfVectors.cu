#include "hip/hip_runtime.h"
%%writefile SumOfVectors.cu
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorsadd(int *x,int *y, int *z)    
{
    int index=blockIdx.x * blockDim.x + threadIdx.x; 
    z[index]=x[index]+y[index];
}

int main()
{
    int n=10;
    int *a, *b, *c;
    int *dev_a,*dev_b,*dev_c;
 
    a = (int *)malloc(n * sizeof(int));
    b = (int *)malloc(n * sizeof(int));
    c = (int *)malloc(n * sizeof(int));
 
    //printf("\n Enter elements of first vector\n");
    for(int i=0;i<n;i++)
    {
        //scanf("%d", &a[i]);
        a[i]=rand()%20;
    }
    //printf("\n Enter elements of second vector\n");
    for(int i=0;i<n;i++)
    {
        //scanf("%d", &b[i]);
        b[i]=rand()%20;
    }

    hipMalloc((void **)&dev_a, n*sizeof(int));
    hipMalloc((void **)&dev_b, n*sizeof(int));
    hipMalloc((void **)&dev_c, n*sizeof(int));

    hipMemcpy(dev_a, a, n*sizeof(int), hipMemcpyHostToDevice);   
    hipMemcpy(dev_b, b, n*sizeof(int), hipMemcpyHostToDevice);
 
    vectorsadd<<<n,1>>>(dev_a, dev_b, dev_c); 
 
    hipMemcpy(c, dev_c, n*sizeof(int), hipMemcpyDeviceToHost);
    
    printf("\nElements of 1st vector:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t", a[i]);
    }

    printf("\nElements of 2nd vector:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t", b[i]);
    }

    printf("\nSum of two vectors:\n ");
    for(int i=0;i<n;i++)
    {
        printf("%d\t", c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
 
    free(a);
    free(b);
    free(c);

    return 0;
}